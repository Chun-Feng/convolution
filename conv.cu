
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>

using namespace std;

// FIXME: the code assumes filter size equals tiling size
#define TILE_X 5
#define TILE_Y 5

#define CUDA_CHECK(ret) do { \
	int errorcode = (ret); \
	if (errorcode != hipSuccess) { \
		std::cout << "cuda error at file " << __FILE__ << " line " << __LINE__ \
			<< ":" << errorcode << std::endl; \
		exit(1); \
	}} while(0)

struct ConvolutionArguments
{
	float *images;
	int image_count;
	int image_width;
	int image_height;
	int image_features;

	float *filters;
	int filter_width;
	int filter_height;

	float *outputs;
	int output_features;
};

void convolution_cpu(const ConvolutionArguments &args)
{
	const int image_size = args.image_width * args.image_height;
	const int filter_size = args.filter_width * args.filter_height;
	const int output_size = image_size;

	for (int i_img = 0; i_img < args.image_count; i_img++) {
		for (int i_feat = 0; i_feat < args.image_features; i_feat++) {
			const int image_index = i_img * args.image_features + i_feat;
			const float *image = &args.images[image_index * image_size];
			for (int i_out_feat = 0; i_out_feat < args.output_features; i_out_feat++) {
				const int filter_index = i_feat * args.output_features + i_out_feat;
				const int output_index = i_img * args.output_features + i_out_feat;

				// convolution between one image feature and one filter
				const float *filter = &args.filters[filter_index * filter_size];
				float *output = &args.outputs[output_index * output_size];

				for (int row = 0; row < args.image_height; row++) {
					for (int col = 0; col < args.image_width; col++) {
						float sum = 0.0;
						for (int frow = 0; frow < args.filter_width; frow++) {
							for (int fcol = 0; fcol < args.filter_height; fcol++) {
								int irow = row + frow - args.filter_height / 2;
								int icol = col + fcol - args.filter_width / 2;
								if (irow >= 0 && irow < args.image_height
										&& icol >= 0 && icol < args.image_width) {
									sum += image[irow * args.image_width + icol] *
										filter[frow * args.filter_width + fcol];
								}
							}
						}
						output[row * args.image_width + col] = sum;
					}
				}
			}
		}
	}
}

__global__
void convolution_kernel(const float *images, const float *filters,
		float *outputs, int image_count, int image_width, int image_height,
		int image_features, int filter_width, int filter_height,
		int output_features)
{
	const int image_size = image_width * image_height;
	const int filter_size = filter_width * filter_height;

	const int col = blockIdx.x;
	const int row = blockIdx.y;
	const int i_img = blockIdx.z;

	const float *image = &images[i_img * image_size];

	for (int i_feat = 0; i_feat < image_features; i_feat++) {
		for (int i_out_feat = 0; i_out_feat < output_features; i_out_feat++) {
			const int filter_index = i_feat * output_features + i_out_feat;
			const int output_index = i_img * output_features + i_out_feat;

			const float *filter = &filters[filter_index * filter_size];
			float *output = &outputs[output_index * image_size];

			float sum = 0.0;
			for (int frow = 0; frow < filter_height; frow++) {
				for (int fcol = 0; fcol < filter_width; fcol++) {
					int irow = row + frow - filter_height / 2;
					int icol = col + fcol - filter_width / 2;
					if (irow >= 0 && irow < image_height
							&& icol >= 0 && icol < image_width) {
						sum += image[irow * image_width + icol] *
							filter[frow * filter_width + fcol];
					}
				}
			}
			output[row * image_width + col] = sum;
		}
	}
}

void convolution_gpu(const ConvolutionArguments &args)
{
	const int images_size = args.image_count * args.image_width *
		args.image_height * args.image_features;
	const int filters_size = args.output_features * args.image_features *
		args.filter_width * args.filter_height;
	const int outputs_size = args.image_count * args.image_width *
		args.image_height * args.output_features;

	// allocate device memory
	float *d_images, *d_filters, *d_outputs;
	CUDA_CHECK(hipMalloc(&d_images, sizeof(float) * images_size));
	CUDA_CHECK(hipMalloc(&d_filters, sizeof(float) * filters_size));
	CUDA_CHECK(hipMalloc(&d_outputs, sizeof(float) * outputs_size));

	// copy data to device
	CUDA_CHECK(hipMemcpy(d_images, args.images, sizeof(float) * images_size,
				hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_filters, args.filters, sizeof(float) * filters_size,
				hipMemcpyHostToDevice));

	dim3 dimGrid(args.image_width / TILE_X + 1,
			args.image_height / TILE_Y + 1, args.image_count);
	dim3 dimBlock(TILE_X, TILE_Y, 1);

	convolution_kernel<<<dimGrid, dimBlock>>>(d_images, d_filters, d_outputs,
			args.image_count, args.image_width, args.image_height,
			args.image_features, args.filter_width, args.filter_height,
			args.output_features);

	// copy data from device
	CUDA_CHECK(hipMemcpy(args.outputs, d_outputs, sizeof(float) * outputs_size,
				hipMemcpyDeviceToHost));

	hipFree(&d_images);
	hipFree(&d_filters);
	hipFree(&d_outputs);
}

// returns t2 - t1 in milliseconds
int timespec_diff_ms(timespec& t1, timespec& t2)
{
	return (t2.tv_sec - t1.tv_sec) * 1000 + (t2.tv_nsec - t1.tv_nsec) / 1e6;
}

int main(int argc, char *argv[])
{
	bool gpu = false;

	if (argc == 2 && argv[1][0] == 'g')
		gpu = true;

	// set arguments

	ConvolutionArguments args;

	args.image_count = 32;
	args.image_width = 128;
	args.image_height = 128;
	args.image_features = 3;

	args.filter_width = TILE_X;
	args.filter_height = TILE_Y;

	args.output_features = 32;

	cout << "initializing" << endl;

	// allocate memory
	const int images_size = args.image_count * args.image_width *
		args.image_height * args.image_features;
	const int filters_size = args.output_features * args.image_features *
		args.filter_width * args.filter_height;
	const int outputs_size = args.image_count * args.image_width *
		args.image_height * args.output_features;
	args.images = new float[images_size];
	args.filters = new float[filters_size];
	args.outputs = new float[outputs_size];

	// initialize inputs
	for (int i = 0; i < images_size; i++)
		args.images[i] = (rand() % 100 - 50) / 50.0;
	for (int i = 0; i < filters_size; i++)
		args.filters[i] = (rand() % 100 - 50) / 50.0;

	hipSetDevice(0); // this is used to initialize a GPU context

	timespec time_before;
	clock_gettime(CLOCK_REALTIME, &time_before);

	if (gpu) {
		cout << "running gpu convolution" << endl;
		convolution_gpu(args);
	} else {
		cout << "running cpu convolution" << endl;
		convolution_cpu(args);
	}

	timespec time_after;
	clock_gettime(CLOCK_REALTIME, &time_after);

	double milliseconds = timespec_diff_ms(time_before, time_after);
	cout << "time consumed: " << milliseconds << " ms" << endl;

	// "read" outputs (for valgrind checking)
	volatile int sink;
	for (int i = 0; i < outputs_size; i++)
		sink = args.outputs[i];

	// free memory
	delete[] args.images;
	delete[] args.filters;
	delete[] args.outputs;

	return 0;
}
