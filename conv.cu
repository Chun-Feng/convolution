
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <sys/time.h>

using namespace std;

// adjustable parameters {{{

#define TILE_SIZE 5

#define TEST_IMAGE_COUNT 32
#define TEST_IMAGE_WIDTH 128
#define TEST_IMAGE_HEIGHT 128
#define TEST_IMAGE_FEATURES 3

#define TEST_OUTPUT_FEATURES 32

// FIXME: the code requires filter size equals tiling size
#define TEST_FILTER_SIZE TILE_SIZE

// }}}

#define CUDA_CHECK(ret) do { \
	int errorcode = (ret); \
	if (errorcode != hipSuccess) { \
		std::cout << "cuda error at file " << __FILE__ << " line " << __LINE__ \
			<< ": " << errorcode << std::endl; \
		exit(1); \
	}} while(0)

// returns t2 - t1 in milliseconds
int timespec_diff_ms(timespec& t1, timespec& t2)
{
	return (t2.tv_sec - t1.tv_sec) * 1000 + (t2.tv_nsec - t1.tv_nsec) / 1e6;
}

struct ConvolutionArguments
{
	float *images;
	int image_count;
	int image_width;
	int image_height;
	int image_features;

	float *filters;
	int filter_size;

	float *outputs;
	int output_features;
};

int convolution_cpu(const ConvolutionArguments &args)
{
	const int image_size = args.image_width * args.image_height;
	const int filter_pixels = args.filter_size * args.filter_size;
	const int stride = args.image_count;

	timespec time_begin, time_end;
	clock_gettime(CLOCK_REALTIME, &time_begin);

	for (int i_img = 0; i_img < args.image_count; i_img++) {
		for (int i_feat = 0; i_feat < args.image_features; i_feat++) {
			for (int i_out_feat = 0; i_out_feat < args.output_features; i_out_feat++) {

				// convolution between one image feature and one filter
				for (int row = 0; row < args.image_height; row++) {
					for (int col = 0; col < args.image_width; col++) {
						float sum = 0.0;
						for (int frow = 0; frow < args.filter_size; frow++) {
							for (int fcol = 0; fcol < args.filter_size; fcol++) {
								int findex = frow * args.filter_size + fcol;
								int irow = row + frow - args.filter_size / 2;
								int icol = col + fcol - args.filter_size / 2;
								if (irow >= 0 && irow < args.image_height
										&& icol >= 0 && icol < args.image_width) {
									sum +=
										args.images[i_feat * image_size * stride + (irow * args.image_width + icol) * stride + i_img] *
										args.filters[i_feat * filter_pixels * args.output_features + findex * args.output_features + i_out_feat];
								}
							}
						}
						args.outputs[i_out_feat * image_size * stride + (row * args.image_width + col) * stride + i_img] = sum;
					}
				}
			}
		}
	}

	clock_gettime(CLOCK_REALTIME, &time_end);
	return timespec_diff_ms(time_begin, time_end);
}

template <int tile_x, int tile_y, int image_features>
__global__
void convolution_kernel(const float *images, const float *filters,
		float *outputs, int image_count, int image_width, int image_height,
		int filter_size, int output_features)
{
	const int image_size = image_width * image_height;
	const int filter_pixels = filter_size * filter_size;
	const int stride = image_count;

	const int col = blockIdx.x * tile_x + threadIdx.x;
	const int row = blockIdx.y * tile_y + threadIdx.y;
	const int i_img = blockIdx.z;

	if (col < image_width && row < image_height) {
		for (int i_feat = 0; i_feat < image_features; i_feat++) {
			for (int i_out_feat = 0; i_out_feat < output_features; i_out_feat++) {

				float sum = 0.0;
				for (int frow = 0; frow < filter_size; frow++) {
					for (int fcol = 0; fcol < filter_size; fcol++) {
						const int irow = row + frow - filter_size / 2;
						const int icol = col + fcol - filter_size / 2;
						const int findex = frow * filter_size + fcol;
						if (irow >= 0 && irow < image_height
								&& icol >= 0 && icol < image_width) {
							sum += images[i_feat * image_size * stride + (irow * image_width + icol) * stride + i_img] *
								filters[i_feat * filter_pixels * output_features + findex * output_features + i_out_feat];
						}
					}
				}
				outputs[i_out_feat * image_size * stride + (row * image_width + col) * stride + i_img] = sum;
			}
		}
	}
}

int convolution_gpu(const ConvolutionArguments &args)
{
	const int images_size = args.image_count * args.image_width *
		args.image_height * args.image_features;
	const int filters_size = args.output_features * args.image_features *
		args.filter_size * args.filter_size;
	const int outputs_size = args.image_count * args.image_width *
		args.image_height * args.output_features;

	// allocate device memory
	float *d_images, *d_filters, *d_outputs;
	CUDA_CHECK(hipMalloc(&d_images, sizeof(float) * images_size));
	CUDA_CHECK(hipMalloc(&d_filters, sizeof(float) * filters_size));
	CUDA_CHECK(hipMalloc(&d_outputs, sizeof(float) * outputs_size));

	// copy data to device
	CUDA_CHECK(hipMemcpy(d_images, args.images, sizeof(float) * images_size,
				hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_filters, args.filters, sizeof(float) * filters_size,
				hipMemcpyHostToDevice));

	dim3 dimGrid(args.image_width / TILE_SIZE + 1, args.image_height / TILE_SIZE + 1,
			args.image_count);
	dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);

	timespec time_begin, time_end;
	clock_gettime(CLOCK_REALTIME, &time_begin);

	convolution_kernel<TILE_SIZE, TILE_SIZE, TEST_IMAGE_FEATURES>
		<<<dimGrid, dimBlock>>>(
			d_images, d_filters, d_outputs,
			args.image_count, args.image_width, args.image_height,
			args.filter_size,
			args.output_features);
	hipDeviceSynchronize(); // wait until convolution_kernel is finished

	clock_gettime(CLOCK_REALTIME, &time_end);

	// copy data from device
	CUDA_CHECK(hipMemcpy(args.outputs, d_outputs, sizeof(float) * outputs_size,
				hipMemcpyDeviceToHost));

	hipFree(&d_images);
	hipFree(&d_filters);
	hipFree(&d_outputs);

	return timespec_diff_ms(time_begin, time_end);
}

int main(int argc, char *argv[])
{
	ConvolutionArguments args;

	args.image_count = TEST_IMAGE_COUNT;
	args.image_width = TEST_IMAGE_WIDTH;
	args.image_height = TEST_IMAGE_HEIGHT;
	args.image_features = TEST_IMAGE_FEATURES;
	args.filter_size = TEST_FILTER_SIZE;
	args.output_features = TEST_OUTPUT_FEATURES;

	cout << "initializing" << endl;

	// allocate memory
	const int images_size = args.image_count * args.image_width *
		args.image_height * args.image_features;
	const int filters_size = args.output_features * args.filter_size *
		args.filter_size * args.filter_size;
	const int outputs_size = args.image_count * args.image_width *
		args.image_height * args.output_features;
	float *images = new float[images_size];
	float *filters = new float[filters_size];
	float *outputs_cpu = new float[outputs_size];
	float *outputs_gpu = new float[outputs_size];

	// initialize inputs
	srand(42);
	for (int i = 0; i < images_size; i++)
		images[i] = (rand() % 100 - 50) / 50.0;
	for (int i = 0; i < filters_size; i++)
		filters[i] = (rand() % 100 - 50) / 50.0;

	cout << "image: count=" << args.image_count
		<< " features=" << args.image_features
		<< " width=" << args.image_width
		<< " height=" << args.image_height << endl;
	cout << "filter: count=" << (args.image_features * args.output_features)
		<< " width=" << args.image_width
		<< " height=" << args.image_height << endl;
	cout << "output: count=" << args.image_count
		<< " features=" << args.output_features
		<< " width=" << args.image_width
		<< " height=" << args.image_height << endl;

	hipSetDevice(0); // this is used to initialize a GPU context

	args.images = images;
	args.filters = filters;

	cout << "running cpu convolution" << endl;
	args.outputs = outputs_cpu;
	int duration_cpu = convolution_cpu(args);

	cout << "running gpu convolution" << endl;
	args.outputs = outputs_gpu;
	int duration_gpu = convolution_gpu(args);

	// compare cpu and gpu answers
	float threshold = 0.00001;
	for (int i = 0; i < outputs_size; i++) {
		if (abs(outputs_cpu[i] - outputs_gpu[i]) >= threshold) {
			cout << "error: answers don't match at index " << i << endl;
			cout << "cpu output: " << outputs_cpu[i] << endl;
			cout << "gpu output: " << outputs_gpu[i] << endl;
			exit(1);
		}
	}
	cout << "compare ok" << endl;

	cout << "cpu duration: " << duration_cpu << " ms" << endl;
	cout << "gpu duration: " << duration_gpu << " ms" << endl;

	// "read" outputs (for valgrind checking)
	volatile int sink;
	for (int i = 0; i < outputs_size; i++)
		sink = outputs_cpu[i];

	// free memory
	delete[] images;
	delete[] filters;
	delete[] outputs_cpu;
	delete[] outputs_gpu;

	return 0;
}
